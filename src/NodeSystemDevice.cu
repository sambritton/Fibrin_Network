#include "hip/hip_runtime.h"
/*
* NodeSystemImplDevice.cu
*
* Created on 8/1/2017
* 		Author: SRB
*/


 
//#include <thrust/version.h>

//#include <hip/hip_runtime.h> 
//#include <hip/hip_runtime.h>
#include <thrust/system_error.h>
#include <thrust/binary_search.h>
#include <thrust/reduce.h>
#include <algorithm> 
#include <thrust/replace.h>
#include <thrust/unique.h> 
#include <thrust/gather.h>
#include <ostream> 
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h> 
#include <thrust/transform_reduce.h> 
#include <math.h>  
#include "ForceDiagramStorage.h"
#include "IncrementExternalForceOnDevice.h"
#include "LinkNodesOnDevice.h"
#include "CalculateEquilibrium.h"
#include "WLCSolveOnDevice.h"
#include "TorsionSolveOnDevice.h"
#include "AdvancePositionOnDevice.h"
#include "BucketSchemeOnDevice.h"
#include "DPDParticle.h"
#include "NodeSystemDevice.h"
#include "NodeSystemDeviceFunctors.h"




using namespace thrust::placeholders;

										  
double NodeSystemDevice::solveForcesOnDevice() {
	//std::cout<<"force: " << nodeInfoVecs.nodeForceX[10] <<" " << nodeInfoVecs.nodeForceY[10] <<" "<<  nodeInfoVecs.nodeForceZ[10]<<std::endl;
	//std::cout<<"loc: " << nodeInfoVecs.nodeLocX[10] <<" " << nodeInfoVecs.nodeLocY[10] <<" "<<  nodeInfoVecs.nodeLocZ[10]<<std::endl;
	//////////////////////////////////////////////////////////////////////////////////////////
	//RESET FORCE TO ZERO AT BEGINNING/////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////
	thrust::fill(nodeInfoVecs.nodeForceX.begin(),nodeInfoVecs.nodeForceX.end(),0);
	thrust::fill(nodeInfoVecs.nodeForceY.begin(),nodeInfoVecs.nodeForceY.end(),0);
	thrust::fill(nodeInfoVecs.nodeForceZ.begin(),nodeInfoVecs.nodeForceZ.end(),0);
	
	//std::cout<<"post fill zeros" << std::endl;
	//std::cout<<"force: " << nodeInfoVecs.nodeForceX[10] <<" " << nodeInfoVecs.nodeForceY[10] <<" "<<  nodeInfoVecs.nodeForceZ[10]<<std::endl;
	///////////////////////////////////////////////////////////////////////////////////////
	//RESET FORCE TO ZERO AT BEGINNING////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////////

	try{initDimensionBucketScheme(
			nodeInfoVecs,
			domainParams, 
			auxVecs, 
			generalParams,
			dpdParticleVariables,
			compressionParams);} //reset dimensions before bucketting domain. Possibly replace with larger domain?}
	catch(thrust::system_error &e){std::cerr << "Error initializing buckets: " << e.what() << std::endl; exit(-1);}
	
	try{buildBucketScheme(nodeInfoVecs, domainParams, 
			auxVecs, generalParams, dpdParticleVariables);}
	catch(thrust::system_error &e){std::cerr << "Error building buckets: " << e.what() << std::endl; exit(-1);}
	
	try{extendBucketScheme(nodeInfoVecs, domainParams, auxVecs);}
	catch(thrust::system_error &e){std::cerr << "Error extending buckets: " << e.what() << std::endl; exit(-1);}
	hipDeviceSynchronize();

	/////////////////////////////////////////////////////////////////////////////
	//safety feature begin //////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////
/*	double tempSizes[] = {std::abs(domainParams.maxX),std::abs(domainParams.minX),
					std::abs(domainParams.maxY), std::abs(domainParams.minY),
					std::abs(domainParams.maxY), std::abs(domainParams.minY)};

  	// using default comparison:
  	double tempMax = *std::max_element(tempSizes,tempSizes+6);

	if ( (tempMax > 10 * compressionParams.originalNetworkLength) ||
		(tempMax > 10 * compressionParams.originalNetworkWidth) ) {
		generalParams.runSim = false;
		std::cout<<"safety feature invoked" << std::endl;
	}
	
	//stop sim if length is larger than strain proportion times original length
	if (abs(compressionParams.averageUpperStrain - compressionParams.averageLowerStrain) > (compressionParams.strainProportion * compressionParams.originalNetworkLength)) {
		generalParams.runSim = false;                                                                             
		std::cout<<"stopping sim, maximumsize reached"<<std::endl;
		 
	}*/
	////////////////////////////////////////////////////////////////////////////
	//safety feature end////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////
	
	/////////////////////////////////////////////////////////////////////
	//LINKING BEGIN//////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////	
	//test turning linking off. 
	double addedLinks = generalParams.currentEdgeCount - generalParams.originEdgeCount;
	

	if (generalParams.linking == true) {
		//default is linking turned on, i.e. Linking = false
		try {
			LinkNodesOnDevice(
					nodeInfoVecs,
					wlcInfoVecs,
					auxVecs,
					torsionInfoVecs,
					generalParams);
			}
		catch(thrust::system_error &e) { 

			std::cerr << "Error linking: " << e.what() << std::endl;		
		}
	}
	hipDeviceSynchronize();

	//apply external force. 
	try {				 									
	IncrementExternalForceOnDevice(nodeInfoVecs, 
		generalParams,
		compressionParams,
		domainParams);
		hipDeviceSynchronize();
	 }
	
	catch(thrust::system_error &e){std::cerr << "Error applying Force : " << e.what() << std::endl; exit(-1);}

	//only counts external force on network nodes since force has been reset. 
	try { compressionParams.totalAppliedForce = (thrust::transform_reduce(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.nodeForceX.begin(),
					nodeInfoVecs.nodeForceY.begin(),
					nodeInfoVecs.nodeForceZ.begin())),
			thrust::make_zip_iterator(				
				thrust::make_tuple(
					nodeInfoVecs.nodeForceX.begin(),
					nodeInfoVecs.nodeForceY.begin(),
					nodeInfoVecs.nodeForceZ.begin())) + generalParams.maxNodeCount,
				NormFunctor(), 0.0, thrust::plus<double>() ) ); } 
	catch(thrust::system_error &e){
		std::cerr << "Error reduce in total force applied: " << e.what() << std::endl; exit(-1);}
					
				//std::cout<<"totalApplied Force: "<< compressionParams.totalAppliedForce<<std::endl;
	try { TorsionSolveOnDevice(nodeInfoVecs, torsionInfoVecs, generalParams); 
		hipDeviceSynchronize(); }
	catch(thrust::system_error &e){std::cerr << "Error Torsion: " << e.what() << std::endl; exit(-1);}
	

	try { WLCSolveOnDevice(nodeInfoVecs, wlcInfoVecs, generalParams); 
		hipDeviceSynchronize(); }
	catch(thrust::system_error &e){std::cerr << "Error WLC : " << e.what() << std::endl; exit(-1);}
	
	return 0.0;//compressionParams.currentNetworkLength;
};


void NodeSystemDevice::solveSystemDevice() {
	//(generalParams.magnitudeForce <= generalParams.maxForce)
	//(compressionParams.targetStrain <= generalParams.maxForce)
	
	double lastTime = 0.0;
	storage->updateStorage();//initial position storage
	bool runIters = true;

	//set initial epsilon
	generalParams.epsilon = (1.0) * 
		sqrt(6.0*generalParams.kB * generalParams.temperature * generalParams.dtTemp / generalParams.viscousDamp);
	std::cout<<"new gen eps begin: "<< generalParams.epsilon<<std::endl;

	while (runIters == true) {
		
		generalParams.iterationCounter++;
		generalParams.currentTime += generalParams.dtTemp;
		//std::cout << "current time: " << std::endl;

		double unused = AdvancePositionOnDevice(
			nodeInfoVecs,
		 	generalParams,
			dpdParticleVariables);
		

		if ((generalParams.iterationCounter % 20000) == 0) {
			storage->print_VTK_File();
			//unsigned maxLinked = *( thrust::max_element(wlcInfoVecs.currentNodeEdgeCountVector.begin(),wlcInfoVecs.currentNodeEdgeCountVector.end()) );
			//std::cout<<"maximum neighbors currently: "<< maxLinked<<std::endl;
			//std::cout<<"maximum neighbors: "<< generalParams.maxNeighborCount <<std::endl;
			
		}

		generalParams.currentLength = solveForcesOnDevice(); //resets and solves forces for next time step
		hipDeviceSynchronize();
		if ((generalParams.iterationCounter % 20000) == 0) {
			double currentStrain = (compressionParams.averageUpperStrain - compressionParams.averageLowerStrain) /
			(compressionParams.originAverageUpperStrain - compressionParams.originAverageLowerStrain ) - 1.0;
			if (currentStrain>4.0){
				runIters=false;
			} 
			thrust::transform( 
				thrust::make_zip_iterator(
					thrust::make_tuple( 
						nodeInfoVecs.nodeForceX.begin(),
						nodeInfoVecs.nodeForceY.begin(),
						nodeInfoVecs.nodeForceZ.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.nodeForceX.begin(),
						nodeInfoVecs.nodeForceY.begin(),
						nodeInfoVecs.nodeForceZ.begin())) + generalParams.maxNodeCount,
				nodeInfoVecs.sumForcesOnNode.begin(),//save vector
				NormFunctor()); 
			
			GetStrainParameters(nodeInfoVecs,
				wlcInfoVecs,  
				generalParams,
				domainParams);
			storage->updateTotalStrain();
		}


		double maxVel = *(thrust::max_element(nodeInfoVecs.nodeVelocity.begin(), nodeInfoVecs.nodeVelocity.end()));
		//std::cout<<"maxvelocity: "<< maxVel<< std::endl;

		//difference in time 
 		if (abs(generalParams.currentTime - lastTime) > (generalParams.lagTime)) {
			 //move epsilon. It will be reset 
 
			generalParams.epsilon += 0.01;
			lastTime = generalParams.currentTime; 
			
			std::cout<<"updating epsilon: "<< generalParams.epsilon<<std::endl;

			double addedEdges = generalParams.currentEdgeCount - generalParams.originEdgeCount;
			std::cout<<"added edges: "<< addedEdges <<std::endl;
			std::cout<<"Minz: "<< domainParams.minZ<<std::endl;
			std::cout<<"Maxz: "<< domainParams.maxZ<<std::endl;
			std::cout<<"Miny: "<< domainParams.minY<<std::endl;
			std::cout<<"Maxy: "<< domainParams.maxY<<std::endl;
			std::cout<<"Minx: "<< domainParams.minX<<std::endl;
			std::cout<<"Maxx: "<< domainParams.maxX<<std::endl;
		} 
	
		if (maxVel < generalParams.epsilon) {
		//only reached if dtMax == dtTemp
			//lastTime = generalParams.currentTime;
			
		
			//store sum of all forces on each node. Used in stress calculations
			thrust::transform( 
				thrust::make_zip_iterator(
					thrust::make_tuple( 
						nodeInfoVecs.nodeForceX.begin(),
						nodeInfoVecs.nodeForceY.begin(),
						nodeInfoVecs.nodeForceZ.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.nodeForceX.begin(),
						nodeInfoVecs.nodeForceY.begin(),
						nodeInfoVecs.nodeForceZ.begin())) + generalParams.maxNodeCount,
				nodeInfoVecs.sumForcesOnNode.begin(),//save vector
				NormFunctor()); 
				
			
			
			storage->updateStorage();
			
			generalParams.totalNumberOfEdges += nodeInfoVecs.idEdgesMadeHost.size();
			nodeInfoVecs.idEdgesMadeHost.resize(0);
	
			
			generalParams.epsilon = (1.0) * 
				sqrt(6.0 * generalParams.kB * generalParams.temperature * generalParams.dtTemp / generalParams.viscousDamp);

			std::cout<<"Maximum vel: "<< maxVel <<std::endl;
			std::cout<<"updating epsilon back to original: "<< generalParams.epsilon<<std::endl;
			generalParams.magnitudeForce += generalParams.df;
			std::cout<<"magnitudeForce: "<< generalParams.magnitudeForce<<std::endl; 
			
		}
		///////////////////////////////////////////////////////////////////////////////
		//EQUILIBRIUM END 
		//////////////////////////////////////////////////////////////////////
		
	}

};



NodeSystemDevice::NodeSystemDevice()  {};

void NodeSystemDevice::assignForceDiagramStorage(std::shared_ptr<ForceDiagramStorage> _storage) {
	storage = _storage;
} 

//__host__ __device__
void NodeSystemDevice::initializeSystem(
	thrust::host_vector<bool>& hostIsNodeFixed,
	thrust::host_vector<double>& hostPosX,
	thrust::host_vector<double>& hostPosY,
	thrust::host_vector<double>& hostPosZ,
	thrust::host_vector<unsigned>& hostWLCEdgeLeft,
	thrust::host_vector<unsigned>& hostWLCEdgeRight,
	thrust::host_vector<double>& hostWLCLenZero,

	thrust::host_vector<unsigned>& hostWLCSubEdgeLeft,
	thrust::host_vector<unsigned>& hostWLCSubEdgeRight,
	thrust::host_vector<double>& hostWLCSubLenZero,
	thrust::host_vector<unsigned>& hostSpringDivisionCount,
	thrust::host_vector<unsigned>& hostTorsionIndexLeft,
	thrust::host_vector<unsigned>& hostTorsionIndexCenter,
	thrust::host_vector<unsigned>& hostTorsionIndexRight,
	thrust::host_vector<double>& hostTorsionAngleZero) {
	
	std::cout<< "total Edge Count: "<< generalParams.originEdgeCount << std::endl;
	std::cout << "max num nodes: " << generalParams.maxNodeCount << std::endl;

	nodeInfoVecs.hostOriginalEdgeLeft = hostWLCEdgeLeft;
	nodeInfoVecs.hostOriginalEdgeRight = hostWLCEdgeRight;

	setNodeVecs(//calls initDimensionBucketScheme
		hostIsNodeFixed,
		hostPosX,
		hostPosY,
		hostPosZ,
		hostSpringDivisionCount);
		
	

	setTorsionVecs(
		hostTorsionIndexLeft,
		hostTorsionIndexCenter,
		hostTorsionIndexRight,
		hostTorsionAngleZero);

	setWLCVecs(	hostWLCSubEdgeLeft,
				hostWLCSubEdgeRight,
				hostWLCSubLenZero );

	setExtras();
};


void NodeSystemDevice::setNodeVecs(
	thrust::host_vector<bool>& hostIsNodeFixed,
	thrust::host_vector<double>& hostPosX,
	thrust::host_vector<double>& hostPosY,
	thrust::host_vector<double>& hostPosZ,
	thrust::host_vector<unsigned>& hostSpringDivisionCount) {


	nodeInfoVecs.idEdgesMadeTemp.resize(generalParams.maxNodeCount * generalParams.maxLinksPerIteration);//corresponds to upperAdj vector size plus a single value to hold number of added nodes
	thrust::fill(nodeInfoVecs.idEdgesMadeTemp.begin(), nodeInfoVecs.idEdgesMadeTemp.end(), 0);

	nodeInfoVecs.sumForcesOnNode.resize(generalParams.maxNodeCount);

	nodeInfoVecs.nodeUpperChoiceForStrain.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeLowerChoiceForStrain.resize(generalParams.maxNodeCount);

	nodeInfoVecs.springDivisionCount.resize(generalParams.maxNodeCount);

	
	nodeInfoVecs.prevNodeLocX.resize(generalParams.maxNodeCount);
	nodeInfoVecs.prevNodeLocY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.prevNodeLocZ.resize(generalParams.maxNodeCount);
	nodeInfoVecs.prevNodeVelX.resize(generalParams.maxNodeCount);
	nodeInfoVecs.prevNodeVelY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.prevNodeVelZ.resize(generalParams.maxNodeCount);
	nodeInfoVecs.prevNodeForceX.resize(generalParams.maxNodeCount);
	nodeInfoVecs.prevNodeForceY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.prevNodeForceZ.resize(generalParams.maxNodeCount);

	nodeInfoVecs.nodeVelocity.resize(generalParams.maxNodeCount);
	
	nodeInfoVecs.nodeLocX.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeLocY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeLocZ.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeVelX.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeVelY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeVelZ.resize(generalParams.maxNodeCount);
	
	
	nodeInfoVecs.nodeForceX.resize(generalParams.maxNodeCount); 
	nodeInfoVecs.nodeForceY.resize(generalParams.maxNodeCount);
	nodeInfoVecs.nodeForceZ.resize(generalParams.maxNodeCount);

	nodeInfoVecs.discretizedEdgeStrain.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	nodeInfoVecs.discretizedEdgeAlignment.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	
	//sized larger for input later
	nodeInfoVecs.deviceEdgeLeft.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	nodeInfoVecs.deviceEdgeRight.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);


	thrust::fill(nodeInfoVecs.discretizedEdgeStrain.begin(), nodeInfoVecs.discretizedEdgeStrain.end(),0.0);
	thrust::fill(nodeInfoVecs.deviceEdgeRight.begin(), nodeInfoVecs.deviceEdgeRight.end(), 0);	//fill force and velocity with zeros for computation.
	thrust::fill(nodeInfoVecs.deviceEdgeLeft.begin(), nodeInfoVecs.deviceEdgeLeft.end(), 0);	//fill force and velocity with zeros for computation.
	thrust::fill(nodeInfoVecs.idEdgesMadeTemp.begin(), nodeInfoVecs.idEdgesMadeTemp.end(), 0);
	
	thrust::fill(nodeInfoVecs.sumForcesOnNode.begin(), nodeInfoVecs.sumForcesOnNode.end(), 0);

	thrust::fill(nodeInfoVecs.nodeUpperChoiceForStrain.begin(), 
		nodeInfoVecs.nodeUpperChoiceForStrain.end(),false);
		
	thrust::fill(nodeInfoVecs.nodeLowerChoiceForStrain.begin(), 
		nodeInfoVecs.nodeLowerChoiceForStrain.end(),false);

	thrust::copy(hostSpringDivisionCount.begin(),hostSpringDivisionCount.end(), nodeInfoVecs.springDivisionCount.begin());
 



	thrust::copy(hostPosX.begin(), hostPosX.end(), nodeInfoVecs.prevNodeLocX.begin());
	thrust::copy(hostPosY.begin(), hostPosY.end(), nodeInfoVecs.prevNodeLocY.begin());
	thrust::copy(hostPosZ.begin(), hostPosZ.end(), nodeInfoVecs.prevNodeLocZ.begin());
	thrust::copy(hostPosX.begin(), hostPosX.end(), nodeInfoVecs.nodeLocX.begin());
	thrust::copy(hostPosY.begin(), hostPosY.end(), nodeInfoVecs.nodeLocY.begin());
	thrust::copy(hostPosZ.begin(), hostPosZ.end(), nodeInfoVecs.nodeLocZ.begin());
	
	
	//copy fixed positions
	nodeInfoVecs.isNodeFixed.resize(generalParams.maxNodeCount);
	thrust::fill(nodeInfoVecs.isNodeFixed.begin(), nodeInfoVecs.isNodeFixed.end(), false);

	//now that all the nodes are loaded in, choose the top to apply strain, and fix the bottom
	
	determineBounds();
	
	//at this point all nodes are filled, so we can generate domainParams before seeding dpd particles. 
	initDimensionBucketScheme(
		nodeInfoVecs, 
		domainParams, 
		auxVecs, 
		generalParams, 
		dpdParticleVariables,
		compressionParams); 
	
	//set original parameters for domain. others will be reset as simulation takes place. 
	domainParams.originMinX = domainParams.minX;
	domainParams.originMaxX = domainParams.maxX;
	domainParams.originMinY = domainParams.minY; 
	domainParams.originMaxY = domainParams.maxY;
	domainParams.originMinZ = domainParams.minZ;
	domainParams.originMaxZ = domainParams.maxZ;
	std::cout<< "node count : " <<nodeInfoVecs.nodeLocY.size()<< std::endl;


	auxVecs.bucketKeys.resize(generalParams.maxNodeCount + dpdParticleVariables.particleCount);
	auxVecs.bucketValues.resize(generalParams.maxNodeCount + dpdParticleVariables.particleCount);
	auxVecs.bucketValuesIncludingNeighbor.resize(27 * (generalParams.maxNodeCount + dpdParticleVariables.particleCount));
	auxVecs.bucketKeysExpanded.resize(27 *( generalParams.maxNodeCount + dpdParticleVariables.particleCount));

};

void NodeSystemDevice::determineBounds() {
	//determin z positions of nodes to be pulled and fixed. 
	
	thrust::device_vector<double> zPosTemp;
	zPosTemp.resize(generalParams.maxNodeCount);
	thrust::copy(nodeInfoVecs.nodeLocZ.begin(), nodeInfoVecs.nodeLocZ.end(), zPosTemp.begin());

	//not used
	//pull at least 10% of nodes. 
	unsigned tempNodeAmmount = static_cast<unsigned>( 0.25 * generalParams.maxNodeCount ); //pull 10% of top nodes
	
	//sort in increasing order
	thrust::sort(zPosTemp.begin(), zPosTemp.end(), thrust::less<double>());
	double length = zPosTemp[ zPosTemp.size()-1 ];
	std::cout<<"start end ZposTemp: "<< zPosTemp[0] << " "<< zPosTemp[zPosTemp.size()-1]<<std::endl;
	
	//upperLevelAlt pulls 10% default. Set in main.cpp using input
	if (generalParams.pullPercent >= 1.0) {
		std::cout<<"ERROR PULL PERCENT MUST BE LESS THAN ONE"<<std::endl;;
	}
	double upperLevelAlt = (1.0 - generalParams.pullPercent) * length;


	double lowerLevel = abs (upperLevelAlt - (zPosTemp[zPosTemp.size()-1]));

	std::cout<<"minimal level final choice for strain choice: " << lowerLevel <<std::endl; 
	
	std::cout<<"maximal level final choice for strain choice: " << upperLevelAlt <<std::endl; 
	
	//apply strain only to original nodes and not added edge subdivision nodes. Set top and bottom

	thrust::replace_if(nodeInfoVecs.nodeUpperChoiceForStrain.begin(), nodeInfoVecs.nodeUpperChoiceForStrain.begin() + generalParams.originNodeCount, 
						nodeInfoVecs.nodeLocZ.begin(),  
						IsGreaterThanLevel( upperLevelAlt ), true);
						
	thrust::replace_if(nodeInfoVecs.nodeLowerChoiceForStrain.begin(), nodeInfoVecs.nodeLowerChoiceForStrain.begin() + generalParams.originNodeCount, 
						nodeInfoVecs.nodeLocZ.begin(),  
						IsLessThanLevel( lowerLevel ), true);
		
	generalParams.numUpperStrainNodes = thrust::count_if(nodeInfoVecs.nodeUpperChoiceForStrain.begin(),nodeInfoVecs.nodeUpperChoiceForStrain.end(), IsEqualToOne( ) );
	generalParams.numLowerStrainNodes = thrust::count_if(nodeInfoVecs.nodeLowerChoiceForStrain.begin(),nodeInfoVecs.nodeLowerChoiceForStrain.end(), IsEqualToOne( ) );
	
	std::cout<<"number of nodes pulled for strain: " << generalParams.numLowerStrainNodes + generalParams.numUpperStrainNodes <<std::endl;

	unsigned numFixed = thrust::count_if(nodeInfoVecs.isNodeFixed.begin(),nodeInfoVecs.isNodeFixed.end(), IsEqualToOne() );
	std::cout<<"number of nodes fixed: " << numFixed <<std::endl;
	zPosTemp.resize(0);

}

void NodeSystemDevice::setTorsionVecs(
	thrust::host_vector<unsigned>& hostTorsionIndexLeft,
	thrust::host_vector<unsigned>& hostTorsionIndexCenter,
	thrust::host_vector<unsigned>& hostTorsionIndexRight,
	thrust::host_vector<double>& hostTorsionAngleZero) {

	unsigned torsion_factor = 500;

	torsionInfoVecs.leftIndex.resize(torsion_factor * generalParams.totalTorsionCount);
	torsionInfoVecs.centerIndex.resize(torsion_factor * generalParams.totalTorsionCount);
	torsionInfoVecs.rightIndex.resize(torsion_factor * generalParams.totalTorsionCount);
	torsionInfoVecs.angleZero.resize(torsion_factor * generalParams.totalTorsionCount);

	thrust::fill(torsionInfoVecs.leftIndex.begin(),torsionInfoVecs.leftIndex.end(),ULONG_MAX);
	thrust::fill(torsionInfoVecs.centerIndex.begin(),torsionInfoVecs.centerIndex.end(),ULONG_MAX);
	thrust::fill(torsionInfoVecs.rightIndex.begin(),torsionInfoVecs.rightIndex.end(),ULONG_MAX);

	//after default value is set, set the real id's
	thrust::copy(hostTorsionIndexLeft.begin(), hostTorsionIndexLeft.end(), torsionInfoVecs.leftIndex.begin());
	thrust::copy(hostTorsionIndexCenter.begin(), hostTorsionIndexCenter.end(), torsionInfoVecs.centerIndex.begin());
	thrust::copy(hostTorsionIndexRight.begin(), hostTorsionIndexRight.end(), torsionInfoVecs.rightIndex.begin());
	
	thrust::transform( 
		thrust::make_zip_iterator(
			thrust::make_tuple( 
				torsionInfoVecs.leftIndex.begin(),
				torsionInfoVecs.centerIndex.begin(),
				torsionInfoVecs.rightIndex.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				torsionInfoVecs.leftIndex.begin(),
				torsionInfoVecs.centerIndex.begin(),
				torsionInfoVecs.rightIndex.begin())) + generalParams.totalTorsionCount,
			torsionInfoVecs.angleZero.begin(),//save vector
		TorsionAngleFunctor(
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data())));
	
	//		std::cout<<" in NSD device values"<<std::endl;
	for (unsigned i = 0; i<generalParams.totalTorsionCount; i++) {
		unsigned n0 = torsionInfoVecs.leftIndex[i];
		unsigned n1 = torsionInfoVecs.centerIndex[i];
		unsigned n2 = torsionInfoVecs.rightIndex[i];
		std::cout<< "angle : "<< n0<< " " << n1<< " " << n2<< " " << torsionInfoVecs.angleZero[i]<<std::endl; 
	}  

	//3x bigger since each spring affects 3 nodes. 
	torsionInfoVecs.forceX.resize(torsion_factor * 3 * generalParams.totalTorsionCount);
	torsionInfoVecs.forceY.resize(torsion_factor * 3 * generalParams.totalTorsionCount);
	torsionInfoVecs.forceZ.resize(torsion_factor * 3 * generalParams.totalTorsionCount);
	torsionInfoVecs.tempForceX.resize(torsion_factor * 3 * generalParams.totalTorsionCount);
	torsionInfoVecs.tempForceY.resize(torsion_factor * 3 * generalParams.totalTorsionCount);
	torsionInfoVecs.tempForceZ.resize(torsion_factor * 3 * generalParams.totalTorsionCount);


	thrust::fill(torsionInfoVecs.forceX.begin(), torsionInfoVecs.forceX.end(), 0.0);
	thrust::fill(torsionInfoVecs.forceY.begin(), torsionInfoVecs.forceY.end(), 0.0);
	thrust::fill(torsionInfoVecs.forceZ.begin(), torsionInfoVecs.forceZ.end(), 0.0);

	torsionInfoVecs.tempTorIndices.resize(torsion_factor * 3 * generalParams.totalTorsionCount);
	torsionInfoVecs.reducedIds.resize(torsion_factor * 3 * generalParams.totalTorsionCount);

 
};

void NodeSystemDevice::setWLCVecs(
	thrust::host_vector<unsigned>& hostWLCSubEdgeLeft,
	thrust::host_vector<unsigned>& hostWLCSubEdgeRight,
	thrust::host_vector<double>& hostWLCSubLenZero ) {

	wlcInfoVecs.globalNeighbors.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	wlcInfoVecs.currentNodeEdgeCountVector.resize(generalParams.maxNodeCount);

	wlcInfoVecs.lengthZero.resize(generalParams.maxNodeCount * generalParams.maxNeighborCount);
	wlcInfoVecs.numOriginalNeighborsNodeVector.resize(generalParams.maxNodeCount);


	thrust::fill(wlcInfoVecs.globalNeighbors.begin(), wlcInfoVecs.globalNeighbors.end(), ULONG_MAX);
	thrust::fill(wlcInfoVecs.currentNodeEdgeCountVector.begin(), wlcInfoVecs.currentNodeEdgeCountVector.end(),0);
	thrust::fill(wlcInfoVecs.lengthZero.begin(), wlcInfoVecs.lengthZero.end(), 0.0);

			   

	nodeInfoVecs.deviceEdgeLeft = hostWLCSubEdgeLeft;
	nodeInfoVecs.deviceEdgeRight = hostWLCSubEdgeRight;
	//scan through hostAdj and put in device.
	for (unsigned id = 0; id < hostWLCSubLenZero.size(); id++) {
		generalParams.totalNumberOfEdges++;
		 unsigned idL = hostWLCSubEdgeLeft[id];
		 unsigned idR = hostWLCSubEdgeRight[id]; 
		 
		//std::cout<< "linking " << idL << " to " <<idR << std::endl;
		
		 double edgeLen = hostWLCSubLenZero[id];	
				//we use the lengthZero vector to identify edges as well.
				//node id is row, column node is connected to row node.
				
		//add edge for left node 		
		unsigned edgeNumL = wlcInfoVecs.currentNodeEdgeCountVector[idL]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexL = idL*generalParams.maxNeighborCount + edgeNumL;
		wlcInfoVecs.lengthZero[indexL] = edgeLen;
		wlcInfoVecs.globalNeighbors[indexL] = idR;
		(wlcInfoVecs.currentNodeEdgeCountVector[idL])++; //right connects to left
  
		//add edge for right node
		unsigned edgeNumR = wlcInfoVecs.currentNodeEdgeCountVector[idR]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexR = idR*generalParams.maxNeighborCount + edgeNumR;
		wlcInfoVecs.lengthZero[indexR] = edgeLen;
		wlcInfoVecs.globalNeighbors[indexR] = idL;
		(wlcInfoVecs.currentNodeEdgeCountVector[idR])++; //left connects to right
		generalParams.currentEdgeCount += 1; 
	} 
	//at this point currentNodeEdgeCountVector holds the number of edges, copy this to 
	thrust::copy(wlcInfoVecs.currentNodeEdgeCountVector.begin(), wlcInfoVecs.currentNodeEdgeCountVector.end(), wlcInfoVecs.numOriginalNeighborsNodeVector.begin());
};

void NodeSystemDevice::setExtras() {
	compressionParams.originalNetworkLength = domainParams.maxZ; //compression along x compressionParams.axis
	compressionParams.originalNetworkWidth = domainParams.maxX;  //strain along z compressionParams.axis.
};


