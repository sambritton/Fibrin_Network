#include "hip/hip_runtime.h"


#include "NodeSystemDevice.h"
#include "NodeSystemBuilder.h"

#include "NodeSystemDeviceFunctors.h"
#include "ForceDiagramStorage.h"
#include <numeric>

ForceDiagramStorage::ForceDiagramStorage(std::weak_ptr<NodeSystemDevice> a_system,
	std::weak_ptr<NodeSystemBuilder> b_system , const std::string& a_fileName) {
	//std::cout << "FDM constructor" << std::endl;
	system = a_system;
	builder = b_system;
	bn = a_fileName; //this will be used later to open files
	std::ofstream statesOutput(a_fileName + ".sta");
	std::ofstream statesOutputStrain(a_fileName + "_Strain.sta");

	std::shared_ptr<NodeSystemDevice> sysA = system.lock();
	std::shared_ptr<NodeSystemBuilder> sysB = builder.lock();
	
	if ((sysA) && (sysB) ){
		unsigned maxNodeCount = sysA->generalParams.maxNodeCount;
		unsigned maxNeighborCount = sysA->generalParams.maxNeighborCount;
		
		statesOutput << "node_count " << maxNodeCount << '\n';
		statesOutput << "origin_node_count " << sysA->generalParams.originNodeCount << '\n';
		statesOutput << "origin_link_count " << sysA->generalParams.originLinkCount << '\n';
		statesOutput << "sub_node_count " << sysA->generalParams.subNodeCount << std::endl;//system->getSubNodesSize() << '\n';
		statesOutput << "link_count " << sysA->generalParams.originEdgeCount << '\n';
		
		for (unsigned edge = 0; edge < sysB->hostWLCEdgeLeft.size(); edge++) {
			unsigned idLeft = sysB->hostWLCEdgeLeft[edge];
			unsigned idRight = sysB->hostWLCEdgeRight[edge];
			statesOutput << '\n' << idLeft << ' ' << idRight;
		}

	}


	statesOutput.close();
}

void ForceDiagramStorage::updateStrain() {
	
/*	std::shared_ptr<NodeSystemDevice> sys = system.lock();
	if (sys) {
		
	statesOutputStrain.open(bn + "_Strain.sta", std::ofstream::out | std::ofstream::app);
		statesOutputStrain << "\ntime " << sys->generalParams.currentTime;
		statesOutputStrain << "\nforce " << sys->compressionParams.totalAppliedForce;
		
		statesOutputStrain << "\nupper_XPos " << sys->domainParams.maxX;
		statesOutputStrain << "\nlower_XPos " << sys->domainParams.minX;
		
		statesOutputStrain << "\nupper_YPos " << sys->domainParams.maxY;
		statesOutputStrain << "\nlower_YPos " << sys->domainParams.minY;
		
		statesOutputStrain << "\nupper_ZPosAve " << sys->compressionParams.averageUpperStrain;
		statesOutputStrain << "\nlower_ZPosAve " << sys->compressionParams.averageLowerStrain;
		statesOutputStrain << "\noriginal_extended_percent " << sys->wlcInfoVecs.percentOriginalEdgesExtended;
		statesOutputStrain << "\noriginal_compressed_percent " << sys->wlcInfoVecs.percentOriginalEdgesCompressed;
		statesOutputStrain << "\noriginal_average_strain " << sys->wlcInfoVecs.averageStrainOriginalEdges;


		for (unsigned i = 0; i < sys->wlcInfoVecs.strainBucketOriginalNeg.size(); i++ ) {
			statesOutputStrain << " \noriginal_strain_neg " << sys->wlcInfoVecs.strainBucketOriginalNeg[i] / (2.0 * sys->generalParams.originEdgeCount);
	
		}		
		for (unsigned i = 0; i < sys->wlcInfoVecs.strainBucketOriginalPos.size(); i++ ) {
			statesOutputStrain << " \noriginal_strain_pos " << sys->wlcInfoVecs.strainBucketOriginalPos[i] /  (2.0 * sys->generalParams.originEdgeCount);
		}
	

 

		statesOutputStrain << "\nadded_extended_percent " << sys->wlcInfoVecs.percentAddedEdgesExtended;
		statesOutputStrain << "\nadded_compressed_percent " << sys->wlcInfoVecs.percentAddedEdgesCompressed;
		statesOutputStrain << "\nadded_average_strain " << sys->wlcInfoVecs.averageStrainAddedEdges;
		
		double sumOfNumsAdded = std::accumulate(sys->wlcInfoVecs.strainBucketAddedNeg.begin(),
			sys->wlcInfoVecs.strainBucketAddedNeg.end(),0.0);
		sumOfNumsAdded += std::accumulate(sys->wlcInfoVecs.strainBucketAddedPos.begin(),
			sys->wlcInfoVecs.strainBucketAddedPos.end(),0.0);
		for (unsigned i = 0; i < sys->wlcInfoVecs.strainBucketAddedNeg.size(); i++ ) {
			statesOutputStrain << " \nadded_strain_neg " << sys->wlcInfoVecs.strainBucketAddedNeg[i]/sumOfNumsAdded;
	
		}		
		for (unsigned i = 0; i < sys->wlcInfoVecs.strainBucketAddedPos.size(); i++ ) {
			statesOutputStrain << " \nadded_strain_pos " << sys->wlcInfoVecs.strainBucketAddedPos[i]/sumOfNumsAdded;
	
		}


		for (unsigned i = 0; i < sys->wlcInfoVecs.alignmentAverage.size(); i++ ) {
			double numEdgesInBin = sys->wlcInfoVecs.numberOfEdgesAlignment[i];
			double val = 0.0;
				
			if (numEdgesInBin != 0.0) {
				val  = sys->wlcInfoVecs.alignmentAverage[i]/numEdgesInBin;
			}
			statesOutputStrain << " \nslice_alignment " << val;
	
		}		
		
		
	} 
	statesOutputStrain.flush();
	statesOutputStrain.close();*/
	
};

void ForceDiagramStorage::updateTotalStrain(void) {
	std::shared_ptr<NodeSystemDevice> sys = system.lock();
	if (sys) {

		double currentStrain = (sys->compressionParams.averageUpperStrain - sys->compressionParams.averageLowerStrain) /
			 (sys->compressionParams.originAverageUpperStrain - sys->compressionParams.originAverageLowerStrain ) - 1.0;
		//first create a new file using the current network strain
		
		std::string format = ".sta";
		std::string strain =  std::to_string(currentStrain);
		std::string initial = "StrainTest/Strain_";
		std::ofstream ofs;
		std::string Filename = initial + strain + format;
		ofs.open(Filename.c_str());



		unsigned maxNeighborCount = sys->generalParams.maxNeighborCount;
		unsigned maxNodeCount = sys->generalParams.maxNodeCount;
		unsigned originalNodeCount = sys->generalParams.originNodeCount;
		unsigned originalEdgeCount = sys->generalParams.originLinkCount;
		unsigned edgeCountDiscretize = sys->generalParams.originEdgeCount;
		//Now first place strain
		ofs << std::setprecision(5) <<std::fixed<< "time " << sys->generalParams.currentTime<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "network_strain " << currentStrain<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "minX " << sys->domainParams.minX<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "maxX " << sys->domainParams.maxX<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "minY " << sys->domainParams.minY<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "maxY " << sys->domainParams.maxY<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "minZ " << sys->domainParams.minX<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "maxZ " << sys->domainParams.maxX<<std::endl;
		
		
		ofs << std::setprecision(5) <<std::fixed<< "total_applied_force " << sys->compressionParams.totalAppliedForce<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "original_node_count " << originalNodeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "node_count_discretize " << maxNodeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "original_edge_count " << originalEdgeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "edge_count_discretize " << edgeCountDiscretize <<std::endl;
		
		//place nodes
		for (unsigned i = 0; i < sys->nodeInfoVecs.nodeLocX.size(); i++) {
			double x = sys->nodeInfoVecs.nodeLocX[i];
			double y = sys->nodeInfoVecs.nodeLocY[i];
			double z = sys->nodeInfoVecs.nodeLocZ[i];
			ofs << std::setprecision(5) <<std::fixed<< "node " << x << " " << y << " " << z <<std::endl;
		
		}
		//place force node is experiencing
		for (unsigned i = 0; i < sys->nodeInfoVecs.nodeLocX.size(); i++) {
			ofs << std::setprecision(5) <<std::fixed<< "force_on_node " << sys->nodeInfoVecs.sumForcesOnNode[i]<<std::endl;
		
		}

		//place original edges
		for (unsigned edge = 0; edge < sys->generalParams.originEdgeCount; edge++) {
			unsigned idL = sys->nodeInfoVecs.deviceEdgeLeft[edge];
			unsigned idR = sys->nodeInfoVecs.deviceEdgeRight[edge];
			ofs <<"original_edge_discretized " <<idL <<" "<< idR <<std::endl;
			
		}
				 
		//place added edges
		for (unsigned edge = sys->generalParams.originEdgeCount; edge < sys->generalParams.currentEdgeCount; edge++) {
			unsigned idL = sys->nodeInfoVecs.deviceEdgeLeft[edge];
			unsigned idR = sys->nodeInfoVecs.deviceEdgeRight[edge];
			ofs <<"added_edge " <<idL <<" "<< idR <<std::endl;
			
		}

		//original edge strain
		for (unsigned i = 0; i < sys->generalParams.originEdgeCount; i++ ){
			double val = sys->nodeInfoVecs.discretizedEdgeStrain[i];

			ofs << std::setprecision(5)<< std::fixed<<"original_edge_strain " << val <<std::endl;
		}
				
		//original edge alignment
		for (unsigned i = 0; i < sys->generalParams.originEdgeCount; i++ ){
			double val = sys->nodeInfoVecs.discretizedEdgeAlignment[i];
			ofs << std::setprecision(5)<< std::fixed<<"original_edge_alignment " << val <<std::endl;
		}

		//added edge strain
		for (unsigned i = sys->generalParams.originEdgeCount; i < sys->generalParams.currentEdgeCount; i++ ){
			double val = sys->nodeInfoVecs.discretizedEdgeStrain[i];
			ofs << std::setprecision(5)<< std::fixed<<"added_edge_strain " << val <<std::endl;
		}
		
		//added links per node.
		for (unsigned i = 0; i < sys->generalParams.maxNodeCount; i++ ){
			unsigned val = sys->wlcInfoVecs.currentNodeEdgeCountVector[i] - 
				sys->wlcInfoVecs.numOriginalNeighborsNodeVector[i];
			ofs << std::setprecision(5)<< std::fixed<<"bind_sites_per_node " << val <<std::endl;
		}



	}
}


void ForceDiagramStorage::print_VTK_File() {
	
	std::shared_ptr<NodeSystemDevice> sys = system.lock();
	if (sys) {	
		iteration+=1;
		unsigned digits = ceil(log10(iteration + 1));
		std::string format = ".vtk";
		std::string Number;
		std::string initial = "AnimationTest/FibrinNetwork_";
		std::ofstream ofs;
		if (digits == 1 || digits == 0) {
			Number = "0000" + std::to_string(iteration);
		}
		else if (digits == 2) {
			Number = "000" + std::to_string(iteration);
		}
		else if (digits == 3) {
			Number = "00" + std::to_string(iteration);
		}
		else if (digits == 4) {
			Number = "0" + std::to_string(iteration);
		}

		std::string Filename = initial + Number + format;

		ofs.open(Filename.c_str());
		
	
		unsigned maxNodeCount = sys->generalParams.maxNodeCount;
		unsigned maxNeighborCount = (sys->generalParams).maxNeighborCount;
		
		unsigned numEdges = sys->generalParams.currentEdgeCount;//sys->nodeInfoVecs.hostEdgeRight.size();
		
		ofs << "# vtk DataFile Version 3.0" << std::endl;
		ofs << "Point representing Sub_cellular elem model" << std::endl;
		ofs << "ASCII" << std::endl << std::endl;
		ofs << "DATASET UNSTRUCTURED_GRID" << std::endl;
		
		 
		ofs << "POINTS " << maxNodeCount << " float" << std::endl;
		for (unsigned i = 0; i< maxNodeCount; i++) {
			double xPos = sys->nodeInfoVecs.nodeLocX[i];
			double yPos = sys->nodeInfoVecs.nodeLocY[i];
			double zPos = sys->nodeInfoVecs.nodeLocZ[i];
			
			ofs << std::setprecision(5) <<std::fixed<< xPos << " " << yPos << " " << zPos << " " << '\n'<< std::fixed;
		}
		//now plot particles

		
		unsigned numCells = numEdges;
		unsigned numNumsInCells = 3 * numEdges;
		
		
		ofs << "CELLS " << numCells << " " << numNumsInCells << std::endl;
		/*for (unsigned i = 0; i< numEdges; i++) {
			unsigned idL = sys->nodeInfoVecs.hostEdgeLeft[i];
			
			unsigned idR = sys->nodeInfoVecs.hostEdgeRight[i];
			ofs<< 2 << " " << idL << " " << idR << std::endl;
		}*/

		
		
		for (unsigned idA = 0; idA < maxNodeCount; idA++ ){
			
			unsigned beginIndex = idA * maxNeighborCount;
			unsigned endIndex = beginIndex + maxNeighborCount;
			for (unsigned i = beginIndex; i < endIndex; i++) {//currentSpringCount is the length of index and value vectors
				unsigned idB = sys->wlcInfoVecs.globalNeighbors[i];//look through possible neighbors. May contain ULONG_MAX
	
				if ((idA < idB) && (idB < maxNodeCount) ) {
					ofs<< 2 << " " << idA << " " << idB << std::endl;
				}
			}
		}
		
		ofs << "CELL_TYPES " << numCells << std::endl;  
		for (unsigned i = 0; i<numEdges; i++) {
			ofs << 3 << std::endl;
		}
		
		
		
		//
		ofs << "CELL_DATA " << numCells << std::endl;
		ofs << "SCALARS magnitude double " << std::endl;
		ofs << "LOOKUP_TABLE default "  << std::endl;
		
		for (unsigned idA = 0; idA < maxNodeCount; idA++ ){
			
			unsigned beginIndex = idA * maxNeighborCount;
			unsigned endIndex = beginIndex + maxNeighborCount;
			for (unsigned i = beginIndex; i < endIndex; i++) {//currentSpringCount is the length of index and value vectors
				unsigned idB = sys->wlcInfoVecs.globalNeighbors[i];//look through possible neighbors. May contain ULONG_MAX
	
				if ((idA < idB) && (idB < maxNodeCount) ) {
					unsigned index = idA * maxNeighborCount + idB;
					double L0 = sys->wlcInfoVecs.lengthZero[i];
					double xL = sys->nodeInfoVecs.nodeLocX[idA];
					double yL = sys->nodeInfoVecs.nodeLocY[idA];
					double zL = sys->nodeInfoVecs.nodeLocZ[idA];
					double xR = sys->nodeInfoVecs.nodeLocX[idB];
					double yR = sys->nodeInfoVecs.nodeLocY[idB];
					double zR = sys->nodeInfoVecs.nodeLocZ[idB];
					
				
					
					double L1 = std::sqrt( (xL - xR)*(xL - xR)+(yL - yR)*(yL - yR)+(zL - zR)*(zL - zR));
					double strain = (L1 - L0) / L0;
					ofs << std::fixed << strain   << std::endl;
				}
			}
		}	

		ofs.close();
	
	}
}

void ForceDiagramStorage::updateStorage() {

	//currentAddedEdges = (system->getDynamicLinks().size() - previousAddedEdges);
	
	std::shared_ptr<NodeSystemDevice> sys = system.lock();
	if (sys) {
		statesOutput.open(bn + ".sta", std::ofstream::out | std::ofstream::app);
		//output.open(bn + ".grm", std::ofstream::out | std::ofstream::app);
		statesOutput << "\nextended percent " << sys->wlcInfoVecs.percentOriginalEdgesExtended;
		statesOutput << "\nforce " << sys->compressionParams.totalAppliedForce;
		statesOutput << "\ntime " << sys->generalParams.currentTime;
		statesOutput << "\nadded edges " << ((sys->nodeInfoVecs.idEdgesMadeHost).size());

		unsigned maxNodeCount = sys->generalParams.maxNodeCount;

		//print new added edges	for current time step recording
		
		for (unsigned i = 0; i < (sys->nodeInfoVecs.idEdgesMadeHost.size()); i++) {
			unsigned idUpper = sys->nodeInfoVecs.idEdgesMadeHost[i];
			if (idUpper != 0) {
				unsigned first = idUpper - maxNodeCount*(idUpper / maxNodeCount); //represents column
				unsigned second = (idUpper / maxNodeCount); //represents row

					statesOutput << '\n' << first << ' ' << second;
			}

		}
		



		for (unsigned i = 0; i < maxNodeCount; ++i) {


			double xPos = sys->nodeInfoVecs.nodeLocX[i];
			double yPos = sys->nodeInfoVecs.nodeLocY[i];
			double zPos = sys->nodeInfoVecs.nodeLocZ[i];
			double xForce = sys->nodeInfoVecs.nodeVelX[i];
			double yForce = sys->nodeInfoVecs.nodeVelY[i];
			double zForce = sys->nodeInfoVecs.nodeVelZ[i];
			double sumOfForces = sys->nodeInfoVecs.sumForcesOnNode[i];
			statesOutput << '\n' << i;

			//auto pos = node->getPosition();
			//auto vel = node->getVelocity();
 
			//for (int k = 0; k < 3; ++k)
			statesOutput << ' ' << xPos << ' ' << yPos << ' ' << zPos;

			//for (int k = 0; k < 3; ++k)
			statesOutput << ' ' << xForce << ' ' << yForce << ' ' << zForce << ' ' << sumOfForces;

		}
	}

	output << magnitudeForce << ' ' << std::endl;
	statesOutput.flush();
	output.flush();
	statesOutput.close();
	output.close();

	std::cout << "*** one step completed ***\n\n";
					 
}




