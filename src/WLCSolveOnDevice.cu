/*
* WLCSolveOnDevice.cu
*
* Created on 8/7/2017 
* 		Author: SRB
*/
#include "NodeSystemDeviceFunctors.h"
#include "NodeSystemDevice.h"
#include "WLCSolveOnDevice.h" 
#include <functional>
#include <algorithm>    // std::transform
#include <vector>     
#include <math.h>  

inline double CVec3_dot(CVec3 v1, CVec3 v2) {
	return (thrust::get<0>(v1)*thrust::get<0>(v2) +
		thrust::get<1>(v1)*thrust::get<1>(v2) +
		thrust::get<2>(v1)*thrust::get<2>(v2));
};
void WLCSolveOnDevice(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,  
	GeneralParams& generalParams) {
 
 
	thrust::counting_iterator<unsigned> startEdgeIter(0);
			  
	//
	thrust::for_each( 
		thrust::make_zip_iterator( 
			thrust::make_tuple(startEdgeIter,
								nodeInfoVecs.springDivisionCount.begin(),
								nodeInfoVecs.isNodeFixed.begin() )),
		thrust::make_zip_iterator(
			thrust::make_tuple(startEdgeIter,
								nodeInfoVecs.springDivisionCount.begin(),
								nodeInfoVecs.isNodeFixed.begin() )) + generalParams.maxNodeCount,
		WLCfunctor(
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeForceX.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeForceY.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.nodeForceZ.data()),
 
			generalParams.kB,
			generalParams.persistenceLengthMon,
			generalParams.CLM,
			generalParams.temperature,
			generalParams.maxNeighborCount,
			generalParams.maxNodeCount,

			thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.springDivisionCount.data()),
			thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
			thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
			thrust::raw_pointer_cast(wlcInfoVecs.numOriginalNeighborsNodeVector.data()) ) );
};

void GetStrainParameters(
	NodeInfoVecs& nodeInfoVecs,
	WLCInfoVecs& wlcInfoVecs,  
	GeneralParams& generalParams,
	DomainParams& domainParams) {
		


		//count positive and negative strains for edges that are not added. If an edge is added, a zero is placed on that strain.
		//notice that each thread will count edges twice, so divide by two at the end
		thrust::counting_iterator<unsigned> startStrainIter(0);

		thrust::fill(nodeInfoVecs.discretizedEdgeStrain.begin(), nodeInfoVecs.discretizedEdgeStrain.end(),0.0);
		thrust::fill(nodeInfoVecs.discretizedEdgeAlignment.begin(), nodeInfoVecs.discretizedEdgeAlignment.end(),0.0);	

		thrust::transform(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.deviceEdgeLeft.begin(),
					nodeInfoVecs.deviceEdgeRight.begin())),
					 
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.deviceEdgeLeft.begin(),
					nodeInfoVecs.deviceEdgeRight.begin())) + generalParams.currentEdgeCount,
					
			//outputs discretized strain etc			
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.discretizedEdgeStrain.begin(),
					nodeInfoVecs.discretizedEdgeAlignment.begin())),
					
			CalculateStrainParamsFunctor(	
				generalParams.originLinkCount,
				generalParams.originEdgeCount,
				generalParams.originNodeCount,
				generalParams.maxNodeCount,
				generalParams.maxNeighborCount,
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocX.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocY.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.nodeLocZ.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.numOriginalNeighborsNodeVector.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.currentNodeEdgeCountVector.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.globalNeighbors.data()),
				thrust::raw_pointer_cast(wlcInfoVecs.lengthZero.data()) ));
			
}; 

  